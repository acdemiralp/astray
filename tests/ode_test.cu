#include "hip/hip_runtime.h"
#include <doctest/doctest.h>

#include <astray/math/ode/ode.hpp>

TEST_CASE("ast::ode")
{
  using method_type  = ast::explicit_method<ast::runge_kutta_4_tableau<float>>;
  using problem_type = ast::initial_value_problem<float, ast::vector3<float>>;
  
  const auto sigma   = 10.0f;
  const auto rho     = 28.0f;
  const auto beta    = 8.0f / 3.0f;
  
  const auto problem = problem_type
  {
    0.0f,                                             /* t0 */
    ast::vector3<float>(16.0f, 16.0f, 16.0f),         /* y0 */
    [&] (const float t, const ast::vector3<float>& y) /* dy/dt = f(t, y) */
    {
      return ast::vector3<float>(sigma * (y[1] - y[0]), y[0] * (rho - y[2]) - y[1], y[0] * y[1] - beta * y[2]);
    }
  };
    
  auto iterator = ast::fixed_step_iterator<method_type, problem_type>(problem, 1.0f /* h */);
  for (auto i = 0; i < 1000; ++i)
    ++iterator;
}