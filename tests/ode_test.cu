#include "hip/hip_runtime.h"
#include <doctest/doctest.h>

#include <astray/math/ode/ode.hpp>

#include <astray/metrics/cartesian/kastor_traschen.hpp>
#include <astray/metrics/cartesian/minkowski.hpp>
#include <astray/metrics/cartesian/goedel.hpp>
#include <astray/metrics/spherical/schwarzschild.hpp>
#include <astray/metrics/boyer_lindquist/kerr.hpp>
#include <astray/core/observer.hpp>
#include <astray/core/geodesic.hpp>
#include <astray/core/ray_tracer.hpp>

TEST_CASE("ast::ode")
{
  using method_type  = ast::explicit_method<ast::runge_kutta_4_tableau<float>>;
  using problem_type = ast::initial_value_problem<float, ast::vector3<float>>;
  
  const auto sigma   = 10.0f;
  const auto rho     = 28.0f;
  const auto beta    = 8.0f / 3.0f;
  
  const auto problem = problem_type
  {
    0.0f,                                             /* t0 */
    ast::vector3<float>(16.0f, 16.0f, 16.0f),         /* y0 */
    [&] (const float t, const ast::vector3<float>& y) /* dy/dt = f(t, y) */
    {
      return ast::vector3<float>(sigma * (y[1] - y[0]), y[0] * (rho - y[2]) - y[1], y[0] * y[1] - beta * y[2]);
    }
  };
    
  auto iterator = ast::fixed_step_iterator<method_type, problem_type> {problem, 1.0f};
  for (auto i = 0; i < 1000; ++i)
    ++iterator;
}