#include <doctest/doctest.h>

#include <astray/math/angle.hpp>
#include <astray/parallel/thrust.hpp>

void range_test()
{
  using type = float;

  thrust::device_vector<type> values     (360);
  std   ::vector       <type> values_host(360);

  for_each(
    make_zip_iterator(make_tuple(thrust::counting_iterator<std::size_t>(0)            , values.begin())),
    make_zip_iterator(make_tuple(thrust::counting_iterator<std::size_t>(values.size()), values.end  ())),
    [ ] __device__ (const auto& iteratee)
    {
      const auto  index = thrust::get<0>(iteratee);
            auto& value = thrust::get<1>(iteratee);

      value = ast::to_radians(static_cast<type>(index));
    });
  
  thrust::copy(values.begin(), values.end(), values_host.begin());
#ifdef __HIPCC__
  hipDeviceSynchronize();
#endif

  for (std::size_t i = 0; i < values_host.size(); ++i)
    REQUIRE(values_host[i] == doctest::Approx(ast::to_radians(static_cast<type>(i))));
  
  for_each(
    values.begin(), 
    values.end  (), 
    [ ] __device__ (auto& iteratee) 
    { 
      iteratee = ast::to_degrees(iteratee); 
    });
  
  thrust::copy(values.begin(), values.end(), values_host.begin());
#ifdef __HIPCC__
  hipDeviceSynchronize();
#endif

  for (std::size_t i = 0; i < values_host.size(); ++i)
    REQUIRE(values_host[i] == doctest::Approx(i));
}

TEST_CASE("ast/angle.hpp")
{
  range_test();
}

